#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   mandel.c                                           :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: abassibe <marvin@42.fr>                    +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/07/28 01:31:18 by abassibe          #+#    #+#             */
/*   Updated: 2017/08/16 04:51:55 by abassibe         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../header/fractol.h"
#include "../cudaheader/cuda_call.h"
/*
static void		mandel_next(t_fract *fract)
{
	long double		zr;
	long double		zi;
	long double		cr;
	long double		ci;
	long double		tmp;

	zr = fract->mdb->zr;
	zi = fract->mdb->zi;
	cr = fract->mdb->cr;
	ci = fract->mdb->ci;
	tmp = fract->mdb->tmp;
	while (zr * zr + zi * zi < 10 && fract->mdb->i < fract->mdb->it_max)
	{
		tmp = zr;
		zr = zr * zr - zi * zi + cr;
		zi = 2 * zi * tmp + ci;
		fract->mdb->i++;
	}
	fract->mdb->zr = zr;
	fract->mdb->zi = zi;

}

void			mandel(t_fract *fract)
{
	int		x;
	int		y;

	x = X - 1;
	y = Y - 1;
	mlx_destroy_image(fract->mlx, fract->vimg);
	fract->vimg = mlx_new_image(fract->mlx, fract->image_x, fract->image_y);
	fract->img = mlx_get_data_addr(fract->vimg, &fract->bpp, &fract->sl, &fract->end);
	while (++x < IMGX + X)
	{
		while (++y < IMGY + Y)
		{
			fract->mdb->cr = x / ZX + X1;
			fract->mdb->ci = y / ZY + Y1;
			fract->mdb->zr = 0;
			fract->mdb->zi = 0;
			fract->mdb->i = 0;
			mandel_next(fract);
			if (fract->mdb->i != fract->mdb->it_max)
				get_color(fract, x - X, y - Y);
		}
		y = Y;
	}
	mlx_put_image_to_window(fract->mlx, fract->win, fract->vimg, 0, 0);
	put_infos(fract);
}
*/
#include <hip/hip_runtime.h>
#include <cuda_call.h>
#include <stdio.h>

__global__ void mandel_next(unsigned int* a, unsigned int constw, unsigned int consth, float middlex, float middley, float scale, unsigned int max)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int index = row * constw + col;
	if(col >= constw || row >= consth) return;
	float ci = (row * scale - middley);
	float cr = (col * scale - middlex) ;
	float x = 0;
	float y = 0;
	float xq = 0;
	float yq = 0;
	unsigned int iteration = 0;
	while (iteration++ < max && (xq + yq) < 4)
	{
		y = 2 * x * y + ci;
		x = xq - yq + cr;
		xq = x * x;
		yq = y * y;
	}
	a[index] = iteration;
}

extern "C" void mandel(unsigned int* a_h, unsigned int constw, unsigned int consth, float middlex, float middley, float scale, unsigned int max, unsigned int reset)
{
	static unsigned int *a_d = NULL;
	static size_t size = 0;
	static dim3 block_size(16, 16);
	static dim3 grid_size(constw / block_size.x + (constw - constw / block_size.x), consth / block_size.y + (consth - consth / block_size.y));
	if (!reset)
	{
		if (size == 0)
		{
			size = constw * consth * sizeof(unsigned int);
			hipMalloc((void **) &a_d, size);
		}
		mandel_next <<< grid_size, block_size, 0 >>> ((unsigned int *)a_d, constw, consth, middlex, middley, scale, max);
		hipMemcpy(a_h, a_d, size, hipMemcpyDeviceToHost);
	}
	else
		hipFree(a_d);
}
